#include "hip/hip_runtime.h"
#include "lodepng.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

//compile with "nvcc GaussianBlur.cu loadpng.cpp -o gaussianBlur" 

__global__ void blurImage(unsigned char * newImage, unsigned char * image,unsigned int width,unsigned int height) {

	        int r = 0;
		int g = 0;
		int b = 0;
		int t = 0;
		int row,col;
		int count = 0;

		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		int pixel = idx*4;

		for(row = (pixel - 4); row<=  (pixel + 4); row+=4){
			// Checking conditions so pixel is available at x
			if ((row > 0) && row < (height * width * 4) && ((row-4)/(4*width) == pixel/(4*width))){
				for(col = (row - (4 * width)); col <=  (row + (4 * width)); col+=(4*width)){
					if(col > 0 && col < (height * width * 4)){
						r += image[col];
						g += image[1+col];
						b += image[2+col]; 
						count++;
					}
				}
			}
		}
		
		t = image[3+pixel];

		newImage[pixel] = r / count;
		newImage[1+pixel] = g / count;
		newImage[2+pixel] = b / count;
		newImage[3+pixel] = t;
}
int time_difference(struct timespec *start,
 struct timespec *finish, 
  long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}
int main(int argc, char **argv){
 struct timespec start, finish;

clock_gettime(CLOCK_MONOTONIC, &start);
clock_gettime(CLOCK_MONOTONIC, &finish);

long long int time_elapsed;
time_difference (&start, &finish, &time_elapsed);
printf("Time elapsed was %lldns or %0.9fs\n", time_elapsed, (time_elapsed/1.0e9));

	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "hck.png";
	const char* newFileName = "output.png";

	lodepng_decode32_file(&image, &width, &height, filename);
	
        printf("Image width = %d height = %d\n", width, height);
	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	blurImage<<<height, width>>>(d_out, d_in, width,height);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
